#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Paralelní programování na GPU (PCG 2022)
 * Projekt c. 1 (cuda)
 * Login: xpolok03
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"
#include "h5Helper.h"

/**
 * Aux function to log last cuda err
 * Author: Robert Crovella, talonmies  Source: https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
 */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv) {
    // Time measurement
    struct timeval t1, t2;

    if (argc != 10) {
        printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
        exit(1);
    }

    // Number of particles
    const int N = std::stoi(argv[1]);
    // Length of time step
    const float dt = std::stof(argv[2]);
    // Number of steps
    const int steps = std::stoi(argv[3]);
    // Number of thread blocks
    const int thr_blc = std::stoi(argv[4]);
    // Write frequency
    int writeFreq = std::stoi(argv[5]);
    // number of reduction threads
    const int red_thr = std::stoi(argv[6]);
    // Number of reduction threads/blocks
    const int red_thr_blc = std::stoi(argv[7]);

    // Size of the simulation CUDA gird - number of blocks
    const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
    // Size of the reduction CUDA grid - number of blocks
    const size_t reductionGrid = (red_thr + red_thr_blc - 1) / red_thr_blc;

    // Log benchmark setup
    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);
    printf("blocks/grid: %lu\n", simulationGrid);
    printf("reduction threads/block: %d\n", red_thr_blc);
    printf("reduction blocks/grid: %lu\n", reductionGrid);

    const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
    writeFreq = (writeFreq > 0) ? writeFreq : 0;


    t_particles particles_cpu;

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                            FILL IN: CPU side memory allocation (step 0)                                          //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


    size_t particles_pos_arr_size = N * sizeof(float4);
    size_t particles_vel_arr_size = N * sizeof(float3);

    // Allocation of pinned, pageable memory at host
    gpuErrchk(hipHostMalloc(&particles_cpu.pos, particles_pos_arr_size));
    gpuErrchk(hipHostMalloc(&particles_cpu.vel, particles_vel_arr_size));


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                              FILL IN: memory layout descriptor (step 0)                                          //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /*
     * Caution! Create only after CPU side allocation
     * parameters:
     *                      Stride of two               Offset of the first
     *  Data pointer        consecutive elements        element in floats,
     *                      in floats, not bytes        not bytes
    */
    MemDesc md(
            &particles_cpu.pos->x, 4, 0,              // Postition in X
            &particles_cpu.pos->y, 4, 0,              // Postition in Y
            &particles_cpu.pos->z, 4, 0,              // Postition in Z
            &particles_cpu.vel->x, 3, 0,              // Velocity in X
            &particles_cpu.vel->y, 3, 0,              // Velocity in Y
            &particles_cpu.vel->z, 3, 0,              // Velocity in Z
            &particles_cpu.pos->w, 4, 0,              // Weight
            N,                                                                  // Number of particles
            recordsNum);                                                              // Number of records in output file

    // Initialisation of helper class and loading of input data
    H5Helper h5Helper(argv[8], argv[9], md);

    try {
        h5Helper.init();
        h5Helper.readParticleData();
    }
    catch (const std::exception &e) {
        std::cerr << e.what() << std::endl;
        return -1;
    }


    t_particles particles_gpu_curr;
    t_particles particles_gpu_next;

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                  FILL IN: GPU side memory allocation (step 0)                                    //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    // Allocation of current and next particles data
    gpuErrchk(hipMalloc<float4>(&particles_gpu_curr.pos, particles_pos_arr_size))
    gpuErrchk(hipMalloc<float3>(&particles_gpu_curr.vel, particles_vel_arr_size))

    gpuErrchk(hipMalloc<float4>(&particles_gpu_next.pos, particles_pos_arr_size))
    gpuErrchk(hipMalloc<float3>(&particles_gpu_next.vel, particles_vel_arr_size))




    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                       FILL IN: memory transfers (step 0)                                         //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Copy input data to gpu memory
    gpuErrchk(hipMemcpy(particles_gpu_curr.pos, particles_cpu.pos, particles_pos_arr_size, hipMemcpyHostToDevice))
    gpuErrchk(hipMemcpy(particles_gpu_curr.vel, particles_cpu.vel, particles_vel_arr_size, hipMemcpyHostToDevice))

    gpuErrchk(hipMemcpy(particles_gpu_next.pos, particles_cpu.pos, particles_vel_arr_size, hipMemcpyHostToDevice))
    gpuErrchk(hipMemcpy(particles_gpu_next.vel, particles_cpu.vel, particles_vel_arr_size, hipMemcpyHostToDevice))


    dim3 dimBlock(thr_blc);
    dim3 dimGrid(simulationGrid);

    // Size of shared memory to keep whole data in current shared mem over block
    size_t sharedMemory = thr_blc * (sizeof(float4) + sizeof(float3));


    float4 *comGPU;
    int *lock;
    gpuErrchk(hipMalloc(&comGPU, sizeof(float4)))
    gpuErrchk(hipMemset(comGPU, 0, sizeof(float4)))
    gpuErrchk(hipMalloc(&lock, sizeof(int)))
    gpuErrchk(hipMemset(lock, 0, sizeof(int)))
    size_t cmo_shared_size = thr_blc * sizeof(float) * 4;

    float4 *comCPU;
    gpuErrchk(hipHostMalloc(&comCPU, sizeof(float4)))

    gettimeofday(&t1, 0);

    size_t record_num = 0;

    // Declare and initialize center of mass and velocity calculation streams and events
    hipStream_t velocity_stream, com_stream;
    hipEvent_t com_updated, com_copied, particles_updated, particles_copied;

    gpuErrchk(hipStreamCreate(&velocity_stream))
    gpuErrchk(hipStreamCreate(&com_stream))

    gpuErrchk(hipEventCreate(&com_updated))
    gpuErrchk(hipEventCreate(&com_copied))
    gpuErrchk(hipEventCreate(&particles_updated))
    gpuErrchk(hipEventCreate(&particles_copied))


    // Replacement of std::swap with indexing mod step -> synchronization over epochs would be necessary with std::swap
    t_particles *particles[2] = {&particles_gpu_curr, &particles_gpu_next};

    for (int s = 0; s < steps; s++) {
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //                                       FILL IN: kernels invocation (step 0)                                     //
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Calculation of t[n+1] particle_data has to wait for t[n] center of mass calculation (3.5.1. IV)
        gpuErrchk(hipStreamWaitEvent(velocity_stream, com_updated))
        calculate_velocity<<<dimGrid, dimBlock, sharedMemory, velocity_stream>>>(*particles[s % 2],
                                                                                 *particles[(s + 1) % 2], N, dt);

        // Calculation of t[n] center of mass has to wait for t[n-1] particle_data calculation (3.5.1. V)
        gpuErrchk(hipStreamWaitEvent(com_stream, particles_updated))

        // Clear center of mass data from last epoch
        gpuErrchk(hipMemsetAsync(comGPU, 0, sizeof(float4), com_stream))
        centerOfMass<<<dimGrid, dimBlock, cmo_shared_size, com_stream>>>(*particles[(s) % 2], &comGPU->x, &comGPU->y,
                                                                         &comGPU->z,
                                                                         &comGPU->w, lock, N);

        // Event to enable start of particle_data calculation for t[n+1] (3.5.1. IV)
        gpuErrchk(hipEventRecord(com_updated, com_stream))


        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //                                          FILL IN: synchronization  (step 4)                                    //
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        if (writeFreq > 0 && (s % writeFreq == 0)) {
            //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            //                          FILL IN: synchronization and file access logic (step 4)                             //
            //////////////////////////////////////////////////////////////////////////////////////////////////////////////////

            // Default synchronization after t[n] particle_data calculation is preserved (3.5.1. III)
            gpuErrchk(hipMemcpyAsync(particles_cpu.pos, (*particles[s % 2]).pos, particles_pos_arr_size,
                                      hipMemcpyDeviceToHost, velocity_stream))
            gpuErrchk(hipMemcpyAsync(particles_cpu.vel, (*particles[s % 2]).vel, particles_vel_arr_size,
                                      hipMemcpyDeviceToHost, velocity_stream))

            // Event to enable writing particle_data to output file (3.5.1. III)
            gpuErrchk(hipEventRecord(particles_copied, velocity_stream))

            // Writing to disk of particle_data t[n] has to wait for t[n-1] particle_data calculation (3.5.1. VI)
            gpuErrchk(hipStreamWaitEvent(velocity_stream, particles_updated))

            // Writing to disk has to wait for copy to finish (3.5.1. VIII)
            // There is no synchronization in following lines, thus copying in parallel with writing to stdout is enabled (3.5.1. VIII)
            gpuErrchk(hipEventSynchronize(particles_copied))
            h5Helper.writeParticleData(record_num);

            // Default synchronization after t[n] center of mass calculation is preserved (3.5.1. VII)
            gpuErrchk(hipMemcpyAsync(comCPU, comGPU, sizeof(float4), hipMemcpyDeviceToHost, com_stream))

            // Event to enable writing particle_data to output file (3.5.1. III)
            gpuErrchk(hipEventRecord(com_copied, com_stream))

            // Writing to disk has to wait for copy to finish (3.5.1. IX -> derived not directly)
            gpuErrchk(hipEventSynchronize(com_copied))
            h5Helper.writeCom(comCPU->x, comCPU->y, comCPU->z, comCPU->w, record_num);

            record_num += 1;
        }

        gpuErrchk(hipEventRecord(particles_updated, velocity_stream));
    }


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //              FILL IN: invocation of center-of-mass kernel (step 3.1, step 3.2, step 4)                           //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    gpuErrchk(hipDeviceSynchronize())

    gpuErrchk(hipPeekAtLastError())

    gpuErrchk(hipMemset(comGPU, 0, sizeof(float4)))

    centerOfMass<<<dimGrid, dimBlock, cmo_shared_size>>>(particles_gpu_curr, &comGPU->x, &comGPU->y, &comGPU->z,
                                                         &comGPU->w, lock, N);


    gettimeofday(&t2, 0);

    // Approximate simulation wall time
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                             FILL IN: memory transfers for particle data (step 0)                                 //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float4 comOnGPU;

    hipMemcpy(particles_cpu.pos, particles_gpu_curr.pos, particles_pos_arr_size, hipMemcpyDeviceToHost);
    hipMemcpy(particles_cpu.vel, particles_gpu_curr.vel, particles_vel_arr_size, hipMemcpyDeviceToHost);


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                         //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipMemcpy(&comOnGPU, comGPU, sizeof(float4), hipMemcpyDeviceToHost);

    gpuErrchk(hipFree(particles_gpu_curr.pos))
    gpuErrchk(hipFree(particles_gpu_curr.vel))
    gpuErrchk(hipFree(particles_gpu_next.pos))
    gpuErrchk(hipFree(particles_gpu_next.vel))
    gpuErrchk(hipFree(comGPU))
    gpuErrchk(hipFree(lock))


    float4 comOnCPU = centerOfMassCPU(md);

    std::cout << "Center of mass on CPU:" << std::endl
              << comOnCPU.x << ", "
              << comOnCPU.y << ", "
              << comOnCPU.z << ", "
              << comOnCPU.w
              << std::endl;

    std::cout << "Center of mass on GPU:" << std::endl
              << comOnGPU.x << ", "
              << comOnGPU.y << ", "
              << comOnGPU.z << ", "
              << comOnGPU.w
              << std::endl;

    // Writing final values to the file
    h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
    h5Helper.writeParticleDataFinal();

    gpuErrchk(hipHostFree(particles_cpu.pos))
    gpuErrchk(hipHostFree(particles_cpu.vel))

    return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
