#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Paralelní programování na GPU (PCG 2022)
 * Projekt c. 1 (cuda)
 * Login: xpolok03
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * Aux function to get size of dynamic shared memory
 * Author: einpoklum  Source: https://stackoverflow.com/questions/42309369/can-my-kernel-code-tell-how-much-shared-memory-it-has-available
 * @return Size of dynamic shared memory
 */
__forceinline__ __device__ unsigned dynamic_smem_size() {
    unsigned ret;
    asm volatile ("mov.u32 %0, %dynamic_smem_size;" : "=r"(ret));
    return ret;
}

/**
 * CUDA kernel to calculate gravitation velocity
 * @param p       - particles
 * @param tmp_vel - temp array for velocities
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_velocity(t_particles p_curr,
                                   t_particles p_next,
                                   int N,
                                   float dt) {

    extern __shared__ float shared[];
    int elements_to_cache = (int) (dynamic_smem_size() / (blockDim.x * sizeof(float)));

    bool is_shared_pos = elements_to_cache >= POS_ELEMENTS;
    bool is_shared_vel = elements_to_cache >= POS_ELEMENTS + VEL_ELEMENTS;
    bool is_mem_pos = !is_shared_pos;
    bool is_mem_vel = !is_shared_vel;

    auto *shared_pos = reinterpret_cast<float4 *>(is_shared_pos ? shared : nullptr);
    auto *shared_vel = reinterpret_cast<float3 *>(is_shared_vel ? &shared[blockDim.x * POS_ELEMENTS] : nullptr);
    unsigned global_id = threadIdx.x + blockIdx.x * blockDim.x;

    float4 *pos_mem = is_shared_pos? shared_pos : p_curr.pos;
    float3 *vel_mem = is_shared_vel? shared_vel : p_curr.vel;

    float4 pos_p1 = p_curr.pos[global_id];
    float3 vel_p1 = p_curr.vel[global_id];
    float4 pos_p2;
    float3 vel_p2;


    float r, dx, dy, dz, r3, Fg_dt_m2_r, weight_difference, weight_sum, double_m2;
    float3 v_temp = {0.0f, 0.0f, 0.0f};
    bool colliding;
    unsigned block_offset;
    unsigned load_index;
    for (int block = 0; block < gridDim.x; block++) {
        block_offset = block * blockDim.x;
        load_index = block_offset + threadIdx.x;
        if (is_shared_pos) {
            shared_pos[threadIdx.x] = p_curr.pos[load_index];
        }
        if (is_shared_vel) {
            shared_vel[threadIdx.x] = p_curr.vel[load_index];
        }
        __syncthreads();
        for (int i = 0; i < blockDim.x; i++) {
            pos_p2 = pos_mem[block_offset *is_mem_pos + i];
            vel_p2 = vel_mem[block_offset *is_mem_vel + i];
            dx = pos_p2.x - pos_p1.x;
            dy = pos_p2.y - pos_p1.y;
            dz = pos_p2.z - pos_p1.z;
            r = sqrt(dx * dx + dy * dy + dz * dz);
            r3 = r * r * r + FLT_MIN;
            colliding = r > 0.0f && r <= COLLISION_DISTANCE;
            pos_p2.w = block_offset + i < N && global_id < N ? pos_p2.w : 0.0f;
            weight_difference = pos_p1.w - pos_p2.w;
            weight_sum = pos_p1.w + pos_p2.w;
            double_m2 = pos_p2.w * 2.0f;

            Fg_dt_m2_r = G * dt / r3 * pos_p2.w;

            v_temp.x += colliding ? ((vel_p1.x * weight_difference + double_m2 * vel_p2.x) / weight_sum) - vel_p1.x
                                  :
                        Fg_dt_m2_r * dx;
            v_temp.y += colliding ? ((vel_p1.y * weight_difference + double_m2 * vel_p2.y) / weight_sum) - vel_p1.y
                                  :
                        Fg_dt_m2_r * dy;
            v_temp.z += colliding ? ((vel_p1.z * weight_difference + double_m2 * vel_p2.z) / weight_sum) - vel_p1.z
                                  :
                        Fg_dt_m2_r * dz;

        }
        __syncthreads();
    }

    if (global_id < N) {
        vel_p1.x += v_temp.x;
        vel_p1.y += v_temp.y;
        vel_p1.z += v_temp.z;
        p_next.vel[global_id] = vel_p1;

        pos_p1.x += vel_p1.x * dt;
        pos_p1.y += vel_p1.y * dt;
        pos_p1.z += vel_p1.z * dt;
        p_next.pos[global_id] = pos_p1;
    }

}// end of calculate_gravitation_velocity
//----------------------------------------------------------------------------------------------------------------------


__global__ void
centerOfMass(t_particles p, float *comX, float *comY, float *comZ, float *comW, int *lock, const int N) {

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc &memDesc) {
    float4 com = {0, 0, 0, 0};

    for (int i = 0; i < memDesc.getDataSize(); i++) {
        // Calculate the vector on the line connecting current body and most recent position of center-of-mass
        const float dx = memDesc.getPosX(i) - com.x;
        const float dy = memDesc.getPosY(i) - com.y;
        const float dz = memDesc.getPosZ(i) - com.z;

        // Calculate weight ratio only if at least one particle isn't massless
        const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                         ? (memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

        // Update position and weight of the center-of-mass according to the weight ration and vector
        com.x += dx * dw;
        com.y += dy * dw;
        com.z += dz * dw;
        com.w += memDesc.getWeight(i);
    }
    return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
