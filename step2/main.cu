#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Paralelní programování na GPU (PCG 2022)
 * Projekt c. 1 (cuda)
 * Login: xpolok03
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"
#include "h5Helper.h"

/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv) {
    // Time measurement
    struct timeval t1, t2;

    if (argc != 10) {
        printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
        exit(1);
    }

    // Number of particles
    const int N = std::stoi(argv[1]);
    // Length of time step
    const float dt = std::stof(argv[2]);
    // Number of steps
    const int steps = std::stoi(argv[3]);
    // Number of thread blocks
    const int thr_blc = std::stoi(argv[4]);
    // Write frequency
    int writeFreq = std::stoi(argv[5]);
    // number of reduction threads
    const int red_thr = std::stoi(argv[6]);
    // Number of reduction threads/blocks
    const int red_thr_blc = std::stoi(argv[7]);

    const int  sharedMemory = std::stoi(argv[8]);

    // Size of the simulation CUDA gird - number of blocks
    const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
    // Size of the reduction CUDA grid - number of blocks
    const size_t reductionGrid = (red_thr + red_thr_blc - 1) / red_thr_blc;

    // Log benchmark setup
    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);
    printf("blocks/grid: %lu\n", simulationGrid);
    printf("reduction threads/block: %d\n", red_thr_blc);
    printf("reduction blocks/grid: %lu\n", reductionGrid);

    const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
    writeFreq = (writeFreq > 0) ? writeFreq : 0;


    t_particles_cpu particles_cpu;

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                            FILL IN: CPU side memory allocation (step 0)                                          //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    size_t particle_arr_size = N * sizeof(float);
    particles_cpu.elements = new float * [N *N_ELEMENTS];

    for (int i = 0; i < N_ELEMENTS; i++) {
        particles_cpu.elements[i] = new float [N];
    }


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                              FILL IN: memory layout descriptor (step 0)                                          //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /*
     * Caution! Create only after CPU side allocation
     * parameters:
     *                      Stride of two               Offset of the first
     *  Data pointer        consecutive elements        element in floats,
     *                      in floats, not bytes        not bytes
    */
    MemDesc md(
            particles_cpu.elements[POS_X], 1, 0,              // Postition in X
            particles_cpu.elements[POS_Y], 1, 0,              // Postition in Y
            particles_cpu.elements[POS_Z], 1, 0,              // Postition in Z
            particles_cpu.elements[VEL_X], 1, 0,              // Velocity in X
            particles_cpu.elements[VEL_Y], 1, 0,              // Velocity in Y
            particles_cpu.elements[VEL_Z], 1, 0,              // Velocity in Z
            particles_cpu.elements[WEIGHT], 1, 0,              // Weight
            N,                                                                  // Number of particles
            recordsNum);                                                        // Number of records in output file

    // Initialisation of helper class and loading of input data
    H5Helper h5Helper(argv[8], argv[9], md);

    try {
        h5Helper.init();
        h5Helper.readParticleData();
    }
    catch (const std::exception &e) {
        std::cerr << e.what() << std::endl;
        return -1;
    }


    t_particles_gpu particles_gpu_curr;
    t_particles_gpu particles_gpu_next;

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                  FILL IN: GPU side memory allocation (step 0)                                    //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    size_t pitch;

    hipMallocPitch<float>(&particles_gpu_curr.elements, &pitch, sizeof(float )  * N_ELEMENTS, N);

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                       FILL IN: memory transfers (step 0)                                         //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    hipMemcpy2D(particles_gpu_curr.elements, pitch, particles_cpu.elements, N_ELEMENTS*sizeof(float), N_ELEMENTS*sizeof(float),
                 N, hipMemcpyHostToDevice);


    dim3 dimBlock(thr_blc);
    dim3 dimGrid(simulationGrid);
    gettimeofday(&t1, 0);
    for (int s = 0; s < steps; s++) {
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //                                       FILL IN: kernels invocation (step 0)                                     //
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        calculate_velocity<<<dimGrid, dimBlock, sharedMemory>>>(particles_gpu_curr, particles_gpu_next, N, dt, pitch, sharedMemory);
        std::swap(particles_gpu_curr, particles_gpu_next);

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //                                          FILL IN: synchronization  (step 4)                                    //
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        if (writeFreq > 0 && (s % writeFreq == 0)) {
            //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            //                          FILL IN: synchronization and file access logic (step 4)                             //
            //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        }
    }


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //              FILL IN: invocation of center-of-mass kernel (step 3.1, step 3.2, step 4)                           //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipDeviceSynchronize();

    gettimeofday(&t2, 0);

    // Approximate simulation wall time
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                             FILL IN: memory transfers for particle data (step 0)                                 //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float4 comOnGPU;
    hipMemcpy2D(particles_cpu.elements, pitch, particles_gpu_curr.elements, N_ELEMENTS*sizeof(float), N_ELEMENTS*sizeof(float),
                 N, hipMemcpyDeviceToHost);


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                         //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float4 comOnCPU = centerOfMassCPU(md);

    std::cout << "Center of mass on CPU:" << std::endl
              << comOnCPU.x << ", "
              << comOnCPU.y << ", "
              << comOnCPU.z << ", "
              << comOnCPU.w
              << std::endl;

    std::cout << "Center of mass on GPU:" << std::endl
              << comOnGPU.x << ", "
              << comOnGPU.y << ", "
              << comOnGPU.z << ", "
              << comOnGPU.w
              << std::endl;

    // Writing final values to the file
    h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
    h5Helper.writeParticleDataFinal();

    return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
